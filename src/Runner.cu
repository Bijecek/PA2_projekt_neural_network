﻿#include <cudaDefs.h>
#include <random>
#include <algorithm>
#include <iostream>

using std::cout;
using std::endl;

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

#include "Layer.cuh"
#include "Loss_functions.cuh"
#include "Activation_functions.cuh"
#include "Back_propagation.cuh"
#include "Datasets.cuh"

int main(int argc, char* argv[])
{
	initializeCUDA(deviceProp);

	// Parametry vstupních dat
	//const int input_size = 4;

	const int input_size = 4;
	const int input_dimension = 2;
	//const int output_size = 4;
	const int output_size = 4;
	const int output_dimension = 1;

	// Hyperparametry
	const int hidden_size = 20;
	// Počet hidden layers
	const int num_hidden = 1;
	// Počet iterací tréninku
	const int n_of_iterations = 50;

	// TODO: learning rate jako konstant memory
	//const float lr = 0.01f;

	// Nahodny generator
	std::mt19937 gen(42);
	std::uniform_real_distribution<float> dist(-1.0, 1.0f);

	//******************************************************************************************|
	//								    INPUT DATASETS										    |						
	//******************************************************************************************|
	
	Dataset ds = getDatasetByName("dataset4");

	const std::vector<float> X = ds.input;
	const std::vector<float> y = ds.target;

	//******************************************************************************************|
	//							   INPUT DATA ALLOCATION ON GPU									|						
	//******************************************************************************************|

	float* d_input;
	checkCudaErrors(hipMalloc(&d_input, input_size * input_dimension * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_input, X.data() , input_size * input_dimension * sizeof(float), hipMemcpyHostToDevice));

	float* d_target;
	checkCudaErrors(hipMalloc(&d_target, output_size * output_dimension * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_target, y.data(), output_size * output_dimension * sizeof(float), hipMemcpyHostToDevice));

	//******************************************************************************************|
	//								      LAYERS APPEND											|						
	//******************************************************************************************|
	std::vector<Layer> layers;

	// Vstupní -> Hidden1
	layers.push_back(createDenseLayer(input_dimension, hidden_size, ActivationFunction::RELU));

	layers.push_back(createDropoutLayer(hidden_size, 0.3f));

	// Hidden1 -> HiddenN
	for (int i = 0; i < num_hidden; i++) {
		layers.push_back(createDenseLayer(hidden_size, hidden_size, ActivationFunction::RELU));
	}

	// HiddenN -> Výstupní
	layers.push_back(createDenseLayer(input_dimension, output_dimension, ActivationFunction::RELU));

	//******************************************************************************************|
	//								  LAYERS ALLOCATION ON GPU								    |					
	//******************************************************************************************|

	// Alokace vrstev
	for (auto& layer : layers) {
		initLayer(layer,input_size);
	}

	//******************************************************************************************|
	//							           OTHER VARIABLES							            |					
	//******************************************************************************************|

	float* d_calculated_loss;
	checkCudaErrors(hipMalloc(&d_calculated_loss, sizeof(float)));

	
	//int h_num_samples = input_size;
	//int d_num_samples;
	//checkCudaErrors(hipMalloc(&d_num_samples, sizeof(int)));
	//checkCudaErrors(hipMemcpy(&d_num_samples, &h_num_samples, sizeof(int), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpyToSymbol((const void*)&num_samples, &input_size, sizeof(int)));

	setNumSamplesConstant(input_size);
	
	// Definice loss pole
	const int gradient_size = output_size * output_dimension;

	float* d_gradient;
	checkCudaErrors(hipMalloc(&d_gradient, gradient_size * sizeof(float)));

	float* h_gradient = new float[gradient_size];


	//******************************************************************************************|
	//								         GRID DEFINITION						            |					
	//******************************************************************************************|

	const int x_tread_count = 16;
	const int y_tread_count = 16;
	//dim3 dimGrid{ 2, 2 ,1 };
	dim3 dimBlock{ 128,1,1 };
	dim3 dimGrid{ 1,1 ,1 };

	//******************************************************************************************|
	//								         MAIN TRAINING LOOP						            |					
	//******************************************************************************************|

	// Hlavní trénovací smyčka
	for (int iteration = 0; iteration < n_of_iterations; iteration++) {
		
		// Resetovani loss
		checkCudaErrors(hipMemset(d_calculated_loss, 0, sizeof(float)));

		// Forward fáze
		float* current_input = d_input;

		for (int i = 0; i < layers.size(); i++) {
			Layer& current_layer = layers[i];

			// Nastavení rozměrů kernelu - dynamicky ho upravujeme podle rozměrů <input_size; layers[i].out>
			dim3 dimBlock{ x_tread_count, y_tread_count ,1 };
				
			// (4 + 16 - 1) / 16
			// (20 + 16 - 1) / 16
			unsigned int x_grid_dim = (input_size+x_tread_count - 1) / x_tread_count;
			unsigned int y_grid_dim = (layers[i].out + y_tread_count - 1) / y_tread_count;

			dim3 dimGrid{
				x_grid_dim,
				y_grid_dim,
				1
			};

			cout << "Kernel executed with: " << x_grid_dim << " " << y_grid_dim << endl;

			if (current_layer.type == LayerType::DENSE) {
				int activation = 0;

				if (current_layer.activation == ActivationFunction::SIGMOID) activation = 1;

				// LOGOVANI
				checkDeviceMatrix<float>(current_layer.activations, input_size * current_layer.out * sizeof(float), 1, input_size * current_layer.out, "%f ", "Before: ");

				forward_pass << <dimGrid, dimBlock >> > (
					current_input,
					current_layer.in,
					current_layer.weights,
					current_layer.biases,
					current_layer.activations,
					current_layer.out,
					activation
					);
			}
			else if (current_layer.type == LayerType::DROPOUT) {
				
				int total = input_size * current_layer.out;
				int blockSize = 256;
				int gridSize = (total + blockSize - 1) / blockSize;

				apply_dropout_forward << <gridSize, blockSize >> > (
					current_input,
					current_layer.mask,
					current_layer.dropout_rate,
					total
				);
			}

			// Změnit vstup
			current_input = current_layer.activations;

			// LOGOVANI
			checkDeviceMatrix<float>(current_layer.activations, input_size *  current_layer.out * sizeof(float), 1, input_size *  current_layer.out, "%f ", "After: ");
		}

		// LOGOVANI - vypis výstupu poslední vrstvy pro všechny vstupy
		checkDeviceMatrix<float>(layers[layers.size() - 1].activations, input_size* layers[layers.size() - 1].out * sizeof(float), 1, input_size* layers[layers.size() - 1].out, "%f ", "Activations: ");

		std::cout << "Forward ok" << std::endl;

		// Počítání loss -- jako vstup je output z předposlední do poslední vrstvy (proto size() - 2)
		compute_loss << <dimGrid, dimBlock >> > (layers[layers.size() - 1].activations, d_target, d_calculated_loss, gradient_size);

		// Přesun loss pole zpátky na host 
		// Mozna zbytecne
		//checkCudaErrors(hipMemcpy(h_output_loss, d_output_loss, compute_loss_size * sizeof(float), hipMemcpyDeviceToHost));
		float* tmp_loss = new float[1];
		checkCudaErrors(hipMemcpy(tmp_loss, d_calculated_loss, sizeof(float), hipMemcpyDeviceToHost));

		// VYPSANI CELKOVE categorical crossentropy LOSS
		cout << "Iteration: " << iteration << " -- loss: " << tmp_loss[0] << std::endl;


		std::cout << "Loss ok" << std::endl;


		compute_gradient << <dimGrid, dimBlock >> > (layers[layers.size() - 1].activations, d_target, d_gradient, gradient_size);
		// LOGOVANI
		//checkDeviceMatrix<float>(d_gradient, gradient_size * sizeof(float), 1, gradient_size, "%f ", "Gradient: ");


		// Copy to GPU
		//checkCudaErrors(hipMemcpy(d_gradient, h_gradient, gradient_size * sizeof(float), hipMemcpyHostToDevice));
		
		
		// Backward fáze
		for (int i = layers.size() - 1; i >= 0; i--) {
			float* input = (i == layers.size() - 1 ? d_gradient : layers[i].gradients);
			float* activation = layers[i].activations;
			int in_size = layers[i].in;
			int out_size = layers[i].out;
			float* weight_matrix = (i == layers.size() - 1) ? nullptr : layers[i+1].weights;
			bool first = (i == layers.size() - 1) ? true : false;
			float* gradient_in = (i == layers.size() - 1) ? nullptr : layers[i+1].gradients;
			float* gradient_out = layers[i].gradients;
			
			int total = in_size * out_size;
			int blockSize = 256;
			int gridSize = (total + blockSize - 1) / blockSize;

			if (layers[i].type == LayerType::DENSE) {
				if (i == layers.size() - 1) {
					backward_pass << <dimGrid, dimBlock >> > (input, activation, in_size, weight_matrix, first, gradient_in, gradient_out, out_size, 0);
				}
				else {
					backward_pass << <dimGrid, dimBlock >> > (input, activation, in_size, weight_matrix, first, gradient_in, gradient_out, out_size, layers[i + 1].out);
				}
			}
			else if (layers[i].type == LayerType::DROPOUT) {

				apply_dropout_backward << <gridSize, blockSize >> > (layers[i].gradients, layers[i].mask, layers[i].dropout_rate, total);
			}

			// LOGOVANI
			//checkDeviceMatrix<float>(layers[i].gradients, input_size * layers[i].out * sizeof(float), 1, input_size * layers[i].out, "%f ", "Gradient calc: ");

			//TODO AKTUALIZACE VAH -- kernel update_parameters
			
			float* input_activations = (i == 0) ? d_input : layers[i-1].activations;
			//int prev_layer_size = (i == 0) ? input_size : layers[i - 1].out;

			update_parameters << <dimGrid, dimBlock >> > (input_activations, layers[i].gradients, layers[i].weights
				, layers[i].biases, layers[i].in, layers[i].out);
			

		}
		for (int i = 0; i < layers.size(); i++) {
			// LOGOVANI
			//checkDeviceMatrix<float>(layers[i].weights, layers[i].in * layers[i].out * sizeof(float), 1, layers[i].in * layers[i].out, "%f ", "Weights: ");
		}

		std::cout << "Backward ok" << std::endl;
		
	}

	cout << "That is all ..." << endl;
}
