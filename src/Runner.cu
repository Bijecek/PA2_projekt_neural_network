#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <random>
#include <algorithm>
#include <iostream>

using std::cout;
using std::endl;

constexpr unsigned int THREADS_PER_BLOCK = 128;

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

struct Layer {
	int in;

	int out;
	// 2D matice jako 1D pole
	float* weights;

	// 1D matice
	float* biases;

	//float* pre_activations;
		
	float* activations;

	float* gradients;
};

// Pomocn� funkce pro v�po�et RELU
__device__ float convert_relu(float sum) {
	return sum > 0.0 ? sum : 0.0;
}

// Dop�edn� pr�chod - jedna vrstva neuronov� s�t�
__global__ void forward(float* input_data, int input_size, float* weight_matrix, float* bias, float* output_data, int output_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < output_size) {
		float sum = 0.0;

		for (int i = 0; i < input_size; i++) {
			sum += input_data[idx * input_size + i] * weight_matrix[idx * input_size + i];
		}
		sum += bias[idx];
		output_data[idx] = convert_relu(sum);
	}
}

//256 velikost -> pracuju s 128 THREADS_PER_BLOCK
__global__ void compute_loss(float* y_predicted, float* y_true, float* loss, int size, float* loss_output) {
	__shared__ float s_loss[THREADS_PER_BLOCK];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int next = THREADS_PER_BLOCK;

	float epsilon = 1e-7;

	if (idx < size) {
		loss_output[idx] = y_true[idx] * logf(fmax(y_predicted[idx], epsilon));
		s_loss[idx] = loss_output[idx];
		__syncthreads();

		while (next > 0) {
			s_loss[idx] += s_loss[idx + next];

			__syncthreads();

			next >>= 1;

			if (idx > next) {
				return;
			}
		}
		if (idx == 0) {
			loss[0] = s_loss[0];
		}
	}
}

// Pomocn� funkce pro derivaci RELU
__device__ float derivate_relu(float sum) {
	return sum > 0 ? 1.0 : 0.0;
}

__global__ void backward(float* input, float* activations, int input_size, float* weight_matrix, bool first, float* gradient_in
	, float* gradient_out, int output_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input_size) {
		if (first) {
			gradient_out[idx] = input[idx] * derivate_relu(activations[idx]);
		}
		else {
			float sum = 0.0;
			for (int i = 0; i < output_size; i++) {
				// Vahova matice je transponovana
				sum += weight_matrix[i * input_size + idx] * gradient_in[i];
			}

			gradient_out[idx] = sum * derivate_relu(activations[idx]);
		}
	}

}
// Funkce pro aktualizaci vah a biasu
__global__ void update_parameters(float* input, float* gradient, float* weight_matrix, float* biases, int input_size, int output_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input_size) {
		float sum = 0.0;
		for (int i = 0; i < output_size; i++) {
			sum += input[i * input_size + idx] * gradient[i];
		}
		// Learning rate
		sum *= 0.05;
		weight_matrix[idx] += sum;



		// Bias TODO
	}
}


int main(int argc, char* argv[])
{
	initializeCUDA(deviceProp);

	// Hyperparametry
	const int input_size = 1024;
	const int hidden_size = 512;
	const int output_size = 1;
	// Po�et hidden layers
	const int num_hidden = 5;
	const int epochs = 100;
	// TODO: learning rate jako konstant memory
	const float lr = 0.01f;

	// Nahodny generator
	std::mt19937 gen(42);
	std::uniform_real_distribution<float> dist(-0.1f, 0.1f);


	std::vector<Layer> layers;

	// Vstupn� -> Hidden1
	layers.push_back({ input_size, hidden_size, nullptr, nullptr, nullptr, nullptr });

	// Hidden1 -> HiddenN
	for (int i = 0; i < num_hidden; i++) {
		layers.push_back({ hidden_size, hidden_size, nullptr, nullptr, nullptr, nullptr });
	}

	// HiddenN -> V�stupn�
	layers.push_back({ hidden_size, output_size, nullptr, nullptr, nullptr, nullptr });

	// Buffery pro vstupn� data
	float* h_input = new float[input_size];
	float* h_target = new float[output_size];

	// N�hodn� inicializace
	// TODO: pou��vat re�ln� data
	for (int i = 0; i < input_size; i++) h_input[i] = dist(gen);
	for (int i = 0; i < output_size; i++) h_target[i] = dist(gen);


	// Alokace vrstev
	for (auto &layer : layers) {
		// V�hov� matice
		checkCudaErrors(hipMalloc(&layer.weights, layer.in * layer.out * sizeof(float)));
		// Bias
		checkCudaErrors(hipMalloc(&layer.biases, layer.out * sizeof(float)));
		// Aktivace
		checkCudaErrors(hipMalloc(&layer.activations, layer.out * sizeof(float)));
		// Gradient
		checkCudaErrors(hipMalloc(&layer.gradients, layer.out * sizeof(float)));

		// Inicializace defaultn�ch hodnot pro v�hy a bias
		// TODO: Generaci p�esunout na GPU
		std::vector<float> temporary_weights(layer.in * layer.out), temporary_biases(layer.out);

		for (auto& v : temporary_weights) v = dist(gen);
		for (auto& v : temporary_biases) v = dist(gen);

		checkCudaErrors(hipMemcpy(layer.weights, temporary_weights.data(), layer.in * layer.out * sizeof(float), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(layer.biases, temporary_biases.data(), layer.out * sizeof(float), hipMemcpyHostToDevice));

	}

	float* d_input;
	checkCudaErrors(hipMalloc(&d_input, input_size * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice));

	float* d_target;
	checkCudaErrors(hipMalloc(&d_target, output_size * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_target, h_target, output_size * sizeof(float), hipMemcpyHostToDevice));


	float* d_loss;
	checkCudaErrors(hipMalloc(&d_loss, sizeof(float)));



	const int n_of_iterations = 10;

	// Velikost v�ech v�stup� -- 1024 * 1
	const int compute_loss_size = input_size * output_size;

	float* d_output_loss;
	checkCudaErrors(hipMalloc(&d_output_loss, compute_loss_size * sizeof(float)));

	float* h_output_loss = new float[compute_loss_size];

	dim3 dimBlock{ THREADS_PER_BLOCK,1,1 };
	dim3 dimGrid{ 1,1,1 };
	// Hlavn� tr�novac� smy�ka

	for (int iteration = 0; iteration < n_of_iterations; iteration++) {

		// Resetovani loss
		checkCudaErrors(hipMemset(d_loss, 0, sizeof(float)));


		// Forward f�ze
		float* current_input = d_input;
		for (int i = 0; i < layers.size(); i++) {
			Layer& current_layer = layers[i];

			forward << <dimGrid, dimBlock >> > (current_input, current_layer.in, current_layer.weights, current_layer.biases,
				current_layer.activations, current_layer.out);

			// Zm�nit vstup
			current_input = current_layer.activations;
		}
		std::cout << "Forward ok" << std::endl;

		// Po��t�n� loss -- jako vstup je output z posledn� vrstvy
		compute_loss << <dimGrid, dimBlock >> > (layers.back().activations, d_target, d_loss, compute_loss_size, d_output_loss);

		// P�esun loss pole zp�tky na host 
		// Mozna zbytecne
		//checkCudaErrors(hipMemcpy(h_output_loss, d_output_loss, compute_loss_size * sizeof(float), hipMemcpyDeviceToHost));
		float* tmp_loss = new float[1];
		checkCudaErrors(hipMemcpy(tmp_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost));

		// VYPSANI CELKOVE LOSS
		cout << "Iteration: " << iteration << " -- loss: " << tmp_loss[0] << std::endl;


		std::cout << "Loss ok" << std::endl;

		// Backward f�ze
		for (int i = layers.size() - 1; i >= 0; i--) {
			float* input = (i == layers.size() - 1 ? d_output_loss : nullptr);
			float* activation = layers[i].activations;
			int in_size = layers[i].in;
			float* weight_matrix = layers[i].weights;
			bool first = (i == layers.size() - 1) ? true : false;
			float* gradient_in = (i == layers.size() - 1) ? nullptr : layers[i + 1].gradients;
			float* gradient_out = layers[i].gradients;
			int out_size = layers[i].out;
			

			backward << <dimGrid, dimBlock >> > (input, activation, in_size, weight_matrix, first, gradient_in, gradient_out, out_size);

			//TODO AKTUALIZACE VAH -- kernel update_parameters
			if (i > 0) {
				float* input_activations = (i == 1) ? d_input : layers[i - 1].activations;
				int prev_layer_size = (i == 1) ? input_size : layers[i - 1].out;

				update_parameters << <dimGrid, dimBlock >> > (input_activations, layers[i].gradients, layers[i].weights
					, layers[i].biases, prev_layer_size, layers[i].out);
			}


		}

		std::cout << "Backward ok" << std::endl;



	}


	cout << "That is all ..." << endl;
}
